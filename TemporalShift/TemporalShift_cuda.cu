#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include "TemporalShift.h"

#include <chrono> // For std::chrono
#include <hip/hip_runtime.h> // For hipDeviceSynchronize
#include <cstdio> // For printf

#define threads  1024


template <typename scalar_t>
__device__ void shift_up_fold(
        scalar_t* __restrict__ input,
        const int n,
        const int s,
        const int c,
        const int h,
        const int w,
        const int bid,
        const int fold
){

    int thread_id = threadIdx.x;
    int stride = blockDim.x;

    int vec_elems = fold * h * w ;

    for (int seg_id = 1; seg_id < s; ++seg_id) {
        scalar_t* input_seg_a = input + bid*s*c*h*w + seg_id *c*h*w ;
        scalar_t* input_seg_b = input + bid*s*c*h*w + (seg_id - 1) *c*h*w ;

        for (int j = thread_id; j < vec_elems; j += stride) {
            input_seg_b[j] = input_seg_a[j];
        }
    }

    scalar_t* input_seg_last = input + bid*s*c*h*w + (s - 1)*c*h*w;
    for (int j = thread_id; j < vec_elems; j += stride) {
        input_seg_last[j] = 0;
    }

}


template <typename scalar_t>
__device__ void shift_down_fold(
        scalar_t* __restrict__ input,
        const int n,
        const int s,
        const int c,
        const int h,
        const int w,
        const int bid,
        const int fold
){

    int thread_id = threadIdx.x;
    int stride = blockDim.x;

    int vec_elems = (fold* h * w) ;

    scalar_t* input_seg_bid = input + bid*s*c*h*w;

    for(int seg_id = s-2; seg_id >= 0; seg_id--){

        scalar_t* input_seg_a = input_seg_bid + seg_id*c*h*w + fold*h*w;
        scalar_t* input_seg_b = input_seg_bid + (seg_id+1)*c*h*w + fold*h*w;

        for (int j = thread_id; j < vec_elems; j += stride) {
            input_seg_b[j] = input_seg_a[j];
        }

    }

    scalar_t* input_seg_last = input_seg_bid + 0*c*h*w + fold*h*w;
    for (int j = thread_id; j < vec_elems; j += stride) {
        input_seg_last[j] = 0;
    }
}
//------------------------------------------//




template <typename scalar_t>
__global__ void temporal_shift_kernel(
        scalar_t* __restrict__ input,
        const int n, //batch_size
        const int s, //segmnet_size
        const int c, //channel_size
        const int h, // image height
        const int w, // image width
        const int fold, // number of chanels to shift up, same number shifted down
        const int forward

) {

    //each block shift fold channels in 1 batch
    if (blockIdx.x >= n ) return;

    const int bid = blockIdx.x ;//batch_id


    if(forward == 1){
        shift_up_fold(input, n,s,c,h,w,bid,fold);
        shift_down_fold(input, n,s,c,h,w,bid,fold);
    }else{
        shift_down_fold(input, n,s,c,h,w,bid,fold);
        shift_up_fold(input, n,s,c,h,w,bid,fold);
    }
}

void temporal_shift_inplace(at::Tensor& input, int64_t fold, int64_t forward) {
    TORCH_CHECK(input.dim() == 5, "Input must be 5D tensor (n,t,c,h,w)");

    const int n = input.size(0);
    const int t = input.size(1);
    const int c = input.size(2);
    const int h = input.size(3);
    const int w = input.size(4);

    const int blocks = (n);


    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "temporal_shift_backward", ([&] {
        temporal_shift_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(), n, t, c, h, w, fold, forward);

        hipDeviceSynchronize();
        auto err = hipGetLastError();
        if (err != hipSuccess) {
            printf("Error in temporal_shift_backward_kernel: %s\n", hipGetErrorString(err));
        }

    }));

}


